#include "hip/hip_runtime.h"
#include <PennesHeatTransfer_gpu.h>
#include<SimulationRessources.h>
#include<chrono>
#define NUMBER_OF_ARRAYS 4
#include<cuda_util.h>
Matrix3D<float>* PennesHeatTransfer_Gpu::Tsaved_h = nullptr;
#if GPU
float *PennesHeatTransfer_Gpu::Tsaved_d = nullptr;

__global__ void cuda_hello(Parameters *_par_d) {//float* _T_d, float* _Ts_d
	printf("Hello World from GPU!\n");
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("x %d, y %d, z %d \n", dim.x, dim.y, dim.z);
	//test_str* _par = (test_str *) _par_d;
	//printf("%f ", (*_par).u);

}
__device__ __host__
float getTd(Parameters* _p, float _T) 
{ 
	//(td_a + td_b * 1e-10 * exp(td_c * 1e-1 * T)) * 1e-6;
	if (_T >= 21 && _T < 200)
	{
		//printf("%f ", exp(_p->td_c * _T));
		return float((_p->td_a + _p->td_b * exp(_p->td_c * _T)) * 1e-6);//
		//return float((1 + _T * _p->td_b) * _p->td_a * 1e-6);
	}

	else
		return float(_p->td_a*1e-6);

}
__global__ void copyT(float* _src, float* _dest, int n)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= n)
		return;

	_dest[tid] = _src[tid];

}
__global__ void setHeatSources(float* _T_d, Parameters* _par_d)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("tid %d ",tid);
	Matrix3D<float> T_m = Matrix3D<float>(_par_d->dim.x, _par_d->dim.y, _par_d->dim.z, _T_d);

	if((int)_par_d->T_heat[tid] != (int) _par_d->defaultT)
		T_m.setValue({ _par_d->heat_x[tid],_par_d->heat_y[tid] ,_par_d->heat_z[tid] }, _par_d->T_heat[tid]);
}
__device__ __host__ void thomas(float* a, float* b, float* c, float* d, int n) {
	/*
	// n is the number of unknowns

	|b0 c0 0 ||x0| |d0|
	|a1 b1 c1||x1|=|d1|
	|0  a2 b2||x2| |d2|

	1st iteration: b0x0 + c0x1 = d0 -> x0 + (c0/b0)x1 = d0/b0 ->

		x0 + g0x1 = r0               where g0 = c0/b0        , r0 = d0/b0

	2nd iteration:     | a1x0 + b1x1   + c1x2 = d1
		from 1st it.: -| a1x0 + a1g0x1        = a1r0
					-----------------------------
						  (b1 - a1g0)x1 + c1x2 = d1 - a1r0

		x1 + g1x2 = r1               where g1=c1/(b1 - a1g0) , r1 = (d1 - a1r0)/(b1 - a1g0)

	3rd iteration:      | a2x1 + b2x2   = d2
		from 2nd it. : -| a2x1 + a2g1x2 = a2r2
					   -----------------------
					   (b2 - a2g1)x2 = d2 - a2r2
		x2 = r2                      where                     r2 = (d2 - a2r2)/(b2 - a2g1)
	Finally we have a triangular matrix:
	|1  g0 0 ||x0| |r0|
	|0  1  g1||x1|=|r1|
	|0  0  1 ||x2| |r2|

	Condition: ||bi|| > ||ai|| + ||ci||

	in this version the c matrix reused instead of g
	and             the d matrix reused instead of r and x matrices to report results
	Written by Keivan Moradi, 2014
	*/
	n--; // since we start from x0 (not x1)
	c[0] /= b[0];
	d[0] /= b[0];

	for (int i = 1; i < n; i++) {
		c[i] /= b[i] - a[i] * c[i - 1];
		d[i] = (d[i] - a[i] * d[i - 1]) / (b[i] - a[i] * c[i - 1]);
	}

	d[n] = (d[n] - a[n] * d[n - 1]) / (b[n] - a[n] * c[n - 1]);

	for (int i = n; i-- > 0;) {

		d[i] -= c[i] * d[i + 1];
	}
}
__global__ void zSweep(float* _T_d, float* _Ts_d, float* _Tss_d, float * _Tnew_d, float* _wb_d, Parameters* _par_d )
{
	extern __shared__ float arrZ[];


	dim3 dim = { _par_d->dim.x, _par_d->dim.y, _par_d->dim.z };
	float defaultValue = _par_d->defaultT;
	float r1, r2, r3;
	float td;
	float dt = _par_d->dt;
	float dx_2[3] = {_par_d->dx_2[0], _par_d->dx_2[1], _par_d->dx_2[2] };


	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= (dim.x * dim.y))
		return;




	float* a = &arrZ[threadIdx.x * NUMBER_OF_ARRAYS * dim.z + 0];
	float* b = &arrZ[threadIdx.x * NUMBER_OF_ARRAYS * dim.z + dim.z];
	float* c = &arrZ[threadIdx.x * NUMBER_OF_ARRAYS * dim.z + dim.z * 2];
	float* d = &arrZ[threadIdx.x * NUMBER_OF_ARRAYS * dim.z + dim.z * 3];

	Matrix3D<float> T_m = Matrix3D<float>(dim.x, dim.y, dim.z, _T_d);
	Matrix3D<float> Ts_m = Matrix3D<float>(dim.x, dim.y, dim.z, _Ts_d);
	Matrix3D<float> Tss_m = Matrix3D<float>(dim.x, dim.y, dim.z, _Tss_d);
	Matrix3D<float> Tnew_m = Matrix3D<float>(dim.x, dim.y, dim.z, _Tnew_d);
	Matrix3D<float> wb_m = Matrix3D<float>(dim.x, dim.y, dim.z, _wb_d);


	

	T_m.setAmbientValue(defaultValue);
	Ts_m.setAmbientValue(defaultValue);
	Tss_m.setAmbientValue(defaultValue);


	int x = tid % dim.x;
	int y = (int)(tid / dim.x);
	
	for (int i = 0; i < dim.z; i++)
	{
		int z = i;
		/*if (x >= _dim.x || x < 0 ||
			y >= _dim.y || y < 0 ||
			z >= _dim.z || z < 0)
			printf("%d , %d , %d \n", x, y, z);*/
		float T_act = (T_m)[{x, y, z}];
		td = getTd(_par_d, T_act);
		
		r1 = r2 = r3 = td * dt / dx_2[2];
		if (r1 < 0)
			printf("r1 %f \n", r1);
		//if ((T_m)[{x, y, z}] > 100 || (T_m)[{x, y, z}] < 0)
		//	printf("Tm %f ", (T_m)[{x, y, z}]);
		//if ((Ts_m)[{x, y, z}] > 100 || (Ts_m)[{x, y, z}] < 0)
		//	printf("Tsm %f ", (Ts_m)[{x, y, z}]);
		//if ((Tss_m)[{x, y, z}] > 100 || (Tss_m)[{x, y, z}] < 0)
		//	printf("Tssm %f \n", (Tss_m)[{x, y, z}]);


		a[i] = -r3 / 2;
		b[i] = 1 + r3;
		c[i] = -r3 / 2;

		d[i] = r1 / 2 * (T_m)[{x - 1, y, z}] + r1 / 2 * (T_m)[{x + 1, y, z}]
			+ r1 / 2 * (Ts_m)[{x - 1, y, z}] + r1 / 2 * (Ts_m)[{x + 1, y, z}]
			+ r2 / 2 * (T_m)[{x, y - 1, z}] + r2 / 2 * (T_m)[{x, y + 1, z}]
			+ r2 / 2 * (Tss_m)[{x, y - 1, z}] + r2 / 2 * (Tss_m)[{x, y + 1, z}]
			+ r3 / 2 * (T_m)[{x, y, z - 1}] + r3 / 2 * (T_m)[{x, y, z + 1}]
			- r1 * (Ts_m)[{x, y, z}] - r2 * (Tss_m)[{x, y, z}]
			+ (1 - r1 - r2 - r3) * (T_m)[{x, y, z}];


		//if (d[i] > 22 || d[i] < 20)
		//{
		//	//printf("falsch  %d  % 0.3f \n" , tid, d[i]);
		//	printf("falsch  %d  % 0.3f, % 0.3f, % 0.3f, % 0.3f, % 0.3f, % 0.3f, % 0.3f, % 0.3f, % 0.3f, % 0.3f, % 0.3f, \n", tid, (T_m)[{x - 1, y, z}], (T_m)[{x + 1, y, z}], (Ts_m)[{x - 1, y, z}], (Ts_m)[{x + 1, y, z}], (T_m)[{x, y - 1, z}], (T_m)[{x, y + 1, z}], (T_m)[{x, y, z - 1}], (T_m)[{x, y, z + 1}], (Ts_m)[{x, y, z}], (T_m)[{x, y, z}], d[i]);
		//}
		////printf(" %.3f ", d[i]);

	}

	a[0] = 0;
	c[dim.z - 1] = 0;



	if (_par_d->boundCond == 0)//neumann -gleiche t_-1 = t_0
	{
		b[0] -= r3 / 2;
		b[dim.z - 1] -= r3 / 2;
	}
	else
	{
		d[0] += r3 / 2 * defaultValue;
		d[dim.z - 1] += r3 / 2 * defaultValue;
	}

	thomas(a, b, c, d, dim.z);
	float P_t;
	for (int i = 0; i < dim.z; i++)
	{

		int z = i;
		//_Tss_d[(tid * _dim.y) + i] = d[i];
		//(Ts_m)[{x, y, z}] = (T_m)[{x, y, z}];
		P_t = (wb_m)[{x, y, z}] * _par_d->cb * (_par_d->Ta - d[z]) * (getTd(_par_d, (T_m)[{x, y, z}]) / _par_d->k);
	//	printf("Pn %.3f wb %.3f \n", P_t, (wb_m)[{x, y, z}]);
		Tnew_m.setValue({ x,y,z }, d[z] + dt * P_t);
		
	}

}
__global__ void ySweep(float* _T_d, float* _Ts_d, float* _Tss_d, Parameters* _par_d)
{
	extern __shared__ float arrY[];

	dim3 dim = { _par_d->dim.x, _par_d->dim.y, _par_d->dim.z };
	float defaultValue = _par_d->defaultT;
	float r1, r2, r3;
	float td;
	float dt = _par_d->dt;
	float dx_2[3] = { _par_d->dx_2[0], _par_d->dx_2[1], _par_d->dx_2[2] };

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= (dim.x * dim.z))
		return;

	
	float* a = &arrY[threadIdx.x * NUMBER_OF_ARRAYS * dim.y + 0];
	float* b = &arrY[threadIdx.x * NUMBER_OF_ARRAYS * dim.y + dim.y];
	float* c = &arrY[threadIdx.x * NUMBER_OF_ARRAYS * dim.y + dim.y * 2];
	float* d = &arrY[threadIdx.x * NUMBER_OF_ARRAYS * dim.y + dim.y * 3];

	Matrix3D<float> T_m = Matrix3D<float>(dim.x, dim.y, dim.z, _T_d);
	Matrix3D<float> Ts_m = Matrix3D<float>(dim.x, dim.y, dim.z, _Ts_d);
	Matrix3D<float> Tss_m = Matrix3D<float>(dim.x, dim.y, dim.z, _Tss_d);
	T_m.setAmbientValue(defaultValue);
	Ts_m.setAmbientValue(defaultValue);



	
	int x = tid % dim.x;
	int z = (int) (tid / dim.x);
	
	for (int i = 0; i < dim.y; i++)
	{
		int y = i;
		/*if (x >= dim.x || x < 0 ||
			y >= dim.y || y < 0 ||
			z >= dim.z || z < 0)
			printf("%d , %d , %d \n", x, y, z);*/

		td = getTd(_par_d, (T_m)[{x, y, z}]);

		r1 = r2 = r3 = td * dt / dx_2[1];

		a[i] = -r2 / 2;
		b[i] = 1 + r2;
		c[i] = -r2 / 2;

		d[i] = r1 / 2 * (T_m)[{x - 1, y, z}] + r1 / 2 * (T_m)[{x + 1, y, z}]
			+ r1 / 2 * (Ts_m)[{x - 1, y, z}] + r1 / 2 * (Ts_m)[{x + 1, y, z}]
			+ r2 / 2 * (T_m)[{x, y - 1, z}] + r2 / 2 * (T_m)[{x, y + 1, z}]
			+ r3 * (T_m)[{x, y, z - 1}] + r3 * (T_m)[{x, y, z + 1}]
			- r1 * (Ts_m)[{x, y, z}]
			+ (1 - r1 - r2 - 2 * r3) * (T_m)[{x, y, z}];


		//if (d[i] > 22 || d[i] < 20)
		//{
		//	//printf("falsch  %d  % 0.3f \n" , tid, d[i]);
		//	printf("falsch  %d  % 0.3f, % 0.3f, % 0.3f, % 0.3f, % 0.3f, % 0.3f, % 0.3f, % 0.3f, % 0.3f, % 0.3f, % 0.3f, \n", tid, (T_m)[{x - 1, y, z}], (T_m)[{x + 1, y, z}], (Ts_m)[{x - 1, y, z}], (Ts_m)[{x + 1, y, z}], (T_m)[{x, y - 1, z}], (T_m)[{x, y + 1, z}], (T_m)[{x, y, z - 1}], (T_m)[{x, y, z + 1}], (Ts_m)[{x, y, z}], (T_m)[{x, y, z}], d[i]);
		//}
		////printf(" %.3f ", d[i]);

	}

	a[0] = 0;
	c[dim.y - 1] = 0;



	if (_par_d->boundCond == 0)//neumann -gleiche t_-1 = t_0
	{
		b[0] -= r2 / 2;
		b[dim.y - 1] -= r2 / 2;
	}
	else
	{
		d[0] += r2 / 2 * defaultValue;
		d[dim.y - 1] += r2 / 2 * defaultValue;
	}

	thomas(a, b, c, d, dim.y);

	for (int i = 0; i < dim.y; i++)
	{

		int y = i;
		//_Tss_d[(tid * dim.y) + i] = d[i];
		//(Tss_m)[{x, y, z}] = (T_m)[{x, y, z}];
		Tss_m.setValue({ x,y,z }, d[i]);
	/*	if (d[i] > 100 || d[i] < 0)
		{
			printf("falsch Thomas y % 0.3f \n", d[i]);
		}*/
	}

}
__global__ void xSweep(float * _T_d, float* _Ts_d, Parameters* _par_d)
{
	extern __shared__ float arr[];

	dim3 dim = { _par_d->dim.x, _par_d->dim.y, _par_d->dim.z };
	//printf("Hallo %d ,%d , %d ", dim.x, dim.y, dim.z);
	
	float defaultValue = _par_d->defaultT;
	float r1, r2, r3;
	float td;
	float dt = _par_d->dt;
	float dx_2[3] = { _par_d->dx_2[0], _par_d->dx_2[1], _par_d->dx_2[2] };

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= (dim.y * dim.z))
		return;



	float* a = &arr[threadIdx.x * NUMBER_OF_ARRAYS * dim.x + 0];
	float* b = &arr[threadIdx.x * NUMBER_OF_ARRAYS * dim.x + dim.x];
	float* c = &arr[threadIdx.x * NUMBER_OF_ARRAYS * dim.x + dim.x * 2];
	float* d = &arr[threadIdx.x * NUMBER_OF_ARRAYS * dim.x + dim.x * 3];
	
	Matrix3D<float> T_m = Matrix3D<float>(dim.x, dim.y, dim.z,_T_d);
	Matrix3D<float> Ts_m = Matrix3D<float>(dim.x, dim.y, dim.z, _Ts_d);
	T_m.setAmbientValue(defaultValue);
	

	
	//int y = (int)(tid / dim.z);
	//int z = tid % dim.z;
	/*if (tid >= 3600)
		printf("TID %d \n ", tid);*/
	int y = tid % dim.y;
	int z = (int)(tid / dim.y);

	for (int i = 0; i < dim.x; i++)
	{
		
		int x = i;
		/*if (x >= dim.x || x < 0 ||
			y >= dim.y || y < 0 ||
			z >= dim.z || z < 0)
			printf("%d , %d , %d \n", x, y, z);*/
		td = getTd(_par_d, (T_m)[{x, y, z}]);
		
		
		r1 = r2 = r3 = td * dt / dx_2[0];

		a[i] = -r1 / 2;
		b[i] = 1 + r1;
		c[i] = -r1 / 2;
		//printf("td %f, dt %f, dx %f ,r1 %f \n", td, dt, dx_2[0], r1);

		float T_im, T_ip, T_jm, T_jp,T_km, T_kp;

		//d[i] = (T_m)[{x, y, z}];
		d[i] = r1 / 2 * (T_m)[{x - 1, y, z}] + r1 / 2 * (T_m)[{x, y, z}]
		+ r2 * (T_m)[{x, y - 1, z}] + r2 * (T_m)[{x, y + 1, z}]
		+ r3 * (T_m)[{x, y, z - 1}] + r3 * (T_m)[{x, y, z + 1}]
		+ (1 - r1 - 2 * r2 - 2 * r3) * (T_m)[{x, y, z}];
		/*if (d[i] != 21.0)
			printf("ahhhh %.6f \n", d[i]);*/ //das noch rausfinden warum nicht gneau 21
		//if (d[i] > 22 || d[i] < 20)
		//{
		//	printf("falsch  %d  % 0.3f \n" , tid, d[i]);
		//	//printf("falsch  %d  % 0.3f, % 0.3f, % 0.3f, % 0.3f, % 0.3f, % 0.3f, % 0.3f, % 0.3f \n", tid, _T_d[im], _T_d[ip], _T_d[jm], _T_d[jp], _T_d[km], _T_d[kp], _T_d[tid * dim.x + i], d[i]);
		//}
		//printf(" %.3f ", d[i]);
			
	}
	
	a[0] = 0;
	c[dim.x - 1] = 0;

	//printf("snow %0.3f \n ", (Ts_m)[{1, 1, 1}]);
	////(Ts_m).setValue({1, 1, 1},15);
	//Ts_m.setAll2(15.0);
	//printf("set %0.3f \n", (Ts_m)[{1, 1, 1}]);
	//Ts_m.setAll2(0.0);
	if (_par_d->boundCond == 0)//neumann -gleiche t_-1 = t_0
	{
		b[0] -= r1 / 2;
		b[dim.x - 1] -= r1 / 2;
	}
	else
	{
		d[0] += r1 / 2 * defaultValue;
		d[dim.x - 1] += r1 / 2 * defaultValue;
	}

	thomas(a, b, c, d, dim.x);
	
	for (int i = 0; i < dim.x; i++)
	{
		
		int x = i;
		//_Ts_d[(tid * dim.x) + i] = d[i];
		//(Ts_m)[{x, y, z}] = d[i];
		Ts_m.setValue({ x,y,z }, d[i]);
		//if (d[i] > 100 || d[i] < 0)
		//{
		//	printf("falsch Thomas z  %0.3f \n", d[i]);
		//}
	}

	

}
#else
float getTd(Parameters* _p, float _T)
{
	//(td_a + td_b * 1e-10 * exp(td_c * 1e-1 * T)) * 1e-6;
	//(td_a + T * td_b) * 1e-6

	return float(_p->td_a * 1e-6);

}
void thomas(float* a, float* b, float* c, float* d, int n) {
	/*
	// n is the number of unknowns

	|b0 c0 0 ||x0| |d0|
	|a1 b1 c1||x1|=|d1|
	|0  a2 b2||x2| |d2|

	1st iteration: b0x0 + c0x1 = d0 -> x0 + (c0/b0)x1 = d0/b0 ->

		x0 + g0x1 = r0               where g0 = c0/b0        , r0 = d0/b0

	2nd iteration:     | a1x0 + b1x1   + c1x2 = d1
		from 1st it.: -| a1x0 + a1g0x1        = a1r0
					-----------------------------
						  (b1 - a1g0)x1 + c1x2 = d1 - a1r0

		x1 + g1x2 = r1               where g1=c1/(b1 - a1g0) , r1 = (d1 - a1r0)/(b1 - a1g0)

	3rd iteration:      | a2x1 + b2x2   = d2
		from 2nd it. : -| a2x1 + a2g1x2 = a2r2
					   -----------------------
					   (b2 - a2g1)x2 = d2 - a2r2
		x2 = r2                      where                     r2 = (d2 - a2r2)/(b2 - a2g1)
	Finally we have a triangular matrix:
	|1  g0 0 ||x0| |r0|
	|0  1  g1||x1|=|r1|
	|0  0  1 ||x2| |r2|

	Condition: ||bi|| > ||ai|| + ||ci||

	in this version the c matrix reused instead of g
	and             the d matrix reused instead of r and x matrices to report results
	Written by Keivan Moradi, 2014
	*/
	n--; // since we start from x0 (not x1)
	c[0] /= b[0];
	d[0] /= b[0];

	for (int i = 1; i < n; i++) {
		c[i] /= b[i] - a[i] * c[i - 1];
		d[i] = (d[i] - a[i] * d[i - 1]) / (b[i] - a[i] * c[i - 1]);
	}

	d[n] = (d[n] - a[n] * d[n - 1]) / (b[n] - a[n] * c[n - 1]);

	for (int i = n; i-- > 0;) {

		d[i] -= c[i] * d[i + 1];
	}
}
#endif

PennesHeatTransfer_Gpu::PennesHeatTransfer_Gpu()
{
#if GPU
	handle = 0;
	bool ft = InitDevice(handle);
	std::cout <<"initDevice: " << (ft ? "True" : "False") << std::endl;
#endif

}
PennesHeatTransfer_Gpu::PennesHeatTransfer_Gpu(unsigned int *_dim)
{
	par_h = (Parameters*)malloc(sizeof(Parameters));
	par_h->dim = { _dim[0],_dim[1],_dim[2] };
	n = par_h->dim.x * par_h->dim.y * par_h->dim.z;

	std::cout << "param0 " << par_h->dim.x << " " << par_h->dim.y << " " << par_h->dim.z << std::endl;

	
	handle = 0;


	

	T_h = new Matrix3D<float>(par_h->dim.x, par_h->dim.y, par_h->dim.z);
	Tnew_h = new Matrix3D<float>(par_h->dim.x, par_h->dim.y, par_h->dim.z);
	wb_h = new Matrix3D<float>(par_h->dim.x, par_h->dim.y, par_h->dim.z);
	//T_h->setAll2(GlobalVariables::boundaryCondition);
	//setDefaultParameters();
#if GPU
	bool ft = InitDevice(handle);
	std::cout << "initDevice: " << (ft ? "True" : "False") << std::endl;

	checkErrorsCuda(hipMalloc((void**)&T_d, sizeof(float) * n));
	checkErrorsCuda(hipMalloc((void**)&Ts_d, sizeof(float) * n));
	checkErrorsCuda(hipMalloc((void**)&Tss_d, sizeof(float) * n));
	checkErrorsCuda(hipMalloc((void**)&Tnew_d, sizeof(float) * n));

	checkErrorsCuda(hipMalloc((void**)&wb_d, sizeof(float) * n));
	std::cout << "sizeofPara" << sizeof(Parameters) << std::endl;
	checkErrorsCuda(hipMalloc((void**)&par_d, sizeof(Parameters)));


	checkLastCudaError("Kernel launch failed.");
#endif



}
PennesHeatTransfer_Gpu::PennesHeatTransfer_Gpu(const PennesHeatTransfer_Gpu& pht)
{
	par_h = (Parameters*)malloc(sizeof(Parameters));
	//par_h->dim = {pht.par_h->dim.x,pht.par_h->dim.y,pht.par_h->dim.z };
	(*par_h) = Parameters((*pht.par_h));
	n = par_h->dim.x * par_h->dim.y * par_h->dim.z;
	

	std::cout << "param0 " << par_h->dim.x << " " << par_h->dim.y << " " << par_h->dim.z << std::endl;


	handle = 0;


	

	T_h = new Matrix3D<float>(par_h->dim.x, par_h->dim.y, par_h->dim.z);
	Tnew_h = new Matrix3D<float>(par_h->dim.x, par_h->dim.y, par_h->dim.z);
	Tsaved_h = pht.Tsaved_h;
	wb_h = new Matrix3D<float>(*pht.wb_h);

#if GPU
	bool ft = InitDevice(handle);
	std::cout << "initDevice: " << (ft ? "True" : "False") << std::endl;
	checkErrorsCuda(hipMalloc((void**)&T_d, sizeof(float) * n));
	checkErrorsCuda(hipMalloc((void**)&Ts_d, sizeof(float) * n));
	checkErrorsCuda(hipMalloc((void**)&Tss_d, sizeof(float) * n));
	checkErrorsCuda(hipMalloc((void**)&Tnew_d, sizeof(float) * n));

	checkErrorsCuda(hipMalloc((void**)&par_d, sizeof(Parameters)));
	checkErrorsCuda(hipMalloc((void**)&wb_d, sizeof(float) * n));

	const int NUM_THREADS_PER_BLOCK = 1024;
	int num_of_threads = n;
	int num_threads_per_block = NUM_THREADS_PER_BLOCK;
	int num_blocks = num_of_threads / num_threads_per_block;
	if (n % num_threads_per_block != 0)
		num_blocks++;

	
	copyT << < num_blocks, num_threads_per_block >> > (pht.wb_d, wb_d, n);
	hipDeviceSynchronize();
	checkLastCudaError("Kernel launch failed.");
#endif
}
PennesHeatTransfer_Gpu::~PennesHeatTransfer_Gpu()
{
#if GPU
	checkErrorsCuda(hipFree(T_d));
	checkErrorsCuda(hipFree(Tnew_d));
	
	checkErrorsCuda(hipFree(Ts_d));
	checkErrorsCuda(hipFree(Tss_d));
	checkErrorsCuda(hipFree(par_d));
	checkErrorsCuda(hipFree(wb_d));
#endif
	delete T_h;
	delete Tnew_h;
	delete Tsaved_h;
	delete wb_h;

}
void PennesHeatTransfer_Gpu::setT_h(Matrix3D<float>* _T_h)
{

	delete T_h;
	T_h = new Matrix3D<float>(*_T_h);
#if	DEBUG
	long float sum = 0;
	for (int i = 0; i < n; i++)
	{
		sum += T_h->getData()[i];
		//std::cout << _Tnew_h[i] << " ";

	}
	printf("T_h: %f avag %f \n", sum, sum / (n));
#endif
#if GPU
	std::chrono::steady_clock::time_point begin_memcpy_malloc = std::chrono::steady_clock::now();
	checkErrorsCuda(hipMemcpy(T_d, T_h->getData(), n * sizeof(float), hipMemcpyHostToDevice));
	std::chrono::steady_clock::time_point end_memcpy_malloc = std::chrono::steady_clock::now();
	std::cout << "memcpy_malloc: " << std::chrono::duration_cast<std::chrono::microseconds>(end_memcpy_malloc - begin_memcpy_malloc).count() << std::endl;
#endif

}
#if GPU
void PennesHeatTransfer_Gpu::setT_d(float* _Tsaved_d)
{
	const int NUM_THREADS_PER_BLOCK = 1024;
	int num_of_threads = n;
	int num_threads_per_block = NUM_THREADS_PER_BLOCK;
	int num_blocks = num_of_threads / num_threads_per_block;
	if (n % num_threads_per_block != 0)
		num_blocks++;

	/*std::cout << "coypTnew2T: num_blocks = " << num_blocks << " :: "
		<< "num_threads_per_block = " << num_threads_per_block << " :: num_of_threads: " << num_of_threads << std::endl;*/

	copyT << < num_blocks, num_threads_per_block >> > (_Tsaved_d, T_d, n);
	hipDeviceSynchronize();
	checkLastCudaError("Kernel launch failed.");

	//Eigentlich nur f�r debug zwecke
	checkErrorsCuda(hipMemcpy(T_h->getData(), T_d, sizeof(float) * n, hipMemcpyDeviceToHost));
	long float sum = 0;

#if DEBUG
	for (int i = 0; i < n; i++)
	{
		sum += T_h->getData()[i];
		//std::cout << _Tnew_h[i] << " ";

	}
	//std::cout <<std::endl<< "SumIs: " << sum << std::endl;// (dim.x * dim.y * dim.z) 
	printf("T_h: %f avag %f \n", sum, sum / (n));
#endif
}
#endif
void PennesHeatTransfer_Gpu::setAll2(float _T) 
{ 
	T_h->setAll2(_T); 
	Tnew_h->setAll2(_T);
#if GPU
	std::chrono::steady_clock::time_point begin_memcpy_malloc = std::chrono::steady_clock::now();
	checkErrorsCuda(hipMemcpy(T_d, T_h->getData(), n * sizeof(float), hipMemcpyHostToDevice));
	checkErrorsCuda(hipMemcpy(Tnew_d, Tnew_h->getData(), n * sizeof(float), hipMemcpyHostToDevice));
	std::chrono::steady_clock::time_point end_memcpy_malloc = std::chrono::steady_clock::now();
	std::cout << "memcpy_malloc: " << std::chrono::duration_cast<std::chrono::microseconds>(end_memcpy_malloc - begin_memcpy_malloc).count() << std::endl;
#endif
}
void PennesHeatTransfer_Gpu::updateTnew_h()
{
	//std::chrono::steady_clock::time_point begin_memcpy = std::chrono::steady_clock::now();
#if GPU	
	checkErrorsCuda(hipMemcpy(Tnew_h->getData(), Tnew_d, sizeof(float) * n, hipMemcpyDeviceToHost));
#endif
	/*std::chrono::steady_clock::time_point end_memcpy = std::chrono::steady_clock::now();
	std::cout << "memcpy: " << std::chrono::duration_cast<std::chrono::microseconds>(end_memcpy - begin_memcpy).count() << std::endl;*/
#if DEBUG
	long float sum = 0;

	for (int i = 0; i < n; i++)
	{
		sum += Tnew_h->getData()[i];
		//std::cout << _Tnew_h[i] << " ";

	}
	//std::cout <<std::endl<< "SumIs: " << sum << std::endl;// (dim.x * dim.y * dim.z) 

	printf("Tnew_h: %f avag %f \n", sum, sum/(n));
#endif

	//return Tnew_h;
}
//void PennesHeatTransfer_Gpu::configTimeSteps(float _dt, int _N = 1)
//{
//
//}
void PennesHeatTransfer_Gpu::saveTimestep()
{
#if GPU
	if (Tsaved_d == nullptr)
	{
		checkErrorsCuda(hipFree(PennesHeatTransfer_Gpu::Tsaved_d));
		checkErrorsCuda(hipMalloc((void**)&Tsaved_d, sizeof(float) * n));
		Tsaved_h = new Matrix3D<float>(par_h->dim.x, par_h->dim.y, par_h->dim.z);
	}


	const int NUM_THREADS_PER_BLOCK = 1024;
	int num_of_threads = n;
	int num_threads_per_block = NUM_THREADS_PER_BLOCK;
	int num_blocks = num_of_threads / num_threads_per_block;
	if (n % num_threads_per_block != 0)
		num_blocks++;


	copyT << < num_blocks, num_threads_per_block >> > (Tnew_d,Tsaved_d, n);
	hipDeviceSynchronize();
	checkLastCudaError("Kernel launch failed.");

	//DEbug
	
	/*std::chrono::steady_clock::time_point end_memcpy = std::chrono::steady_clock::now();
	std::cout << "memcpy: " << std::chrono::duration_cast<std::chrono::microseconds>(end_memcpy - begin_memcpy).count() << std::endl;*/
#else
	if (Tsaved_h == nullptr)
	{
		Tsaved_h = new Matrix3D<float>(par_h->dim.x, par_h->dim.y, par_h->dim.z);
	}
	for (int i = 0; i < n; i++)
		Tsaved_h->setValue(i, Tnew_h->getData()[i]);
#endif

#if DEBUG
#if GPU
	checkErrorsCuda(hipMemcpy(Tsaved_h->getData(), Tsaved_d, sizeof(float) * n, hipMemcpyDeviceToHost));
#endif
	long float sum = 0;

	for (int i = 0; i < n; i++)
	{
		sum += Tsaved_h->getData()[i];
		//std::cout << _Tnew_h[i] << " ";

	}
	//std::cout <<std::endl<< "SumIs: " << sum << std::endl;// (dim.x * dim.y * dim.z) 
	printf("Tsaved_h: %f avag %f \n", sum, sum / (n));
#endif
}
#if GPU
void  PennesHeatTransfer_Gpu::updateParameters()
{
	//std::cout << "param " << par_h->dim.x << " "<< par_h->dim.y << " " << par_h->dim.z << std::endl;

	checkErrorsCuda(hipMemcpy(par_d, par_h, sizeof(Parameters), hipMemcpyHostToDevice));
	//printParameters();
	/*cuda_hello << <1,1>> > (par_d);
	hipDeviceSynchronize();
	checkLastCudaError("Kernel launch failed.");
	std::cout << "hallo" << std::endl;*/

}
#endif
void PennesHeatTransfer_Gpu::setDefaultParameters()
{
	par_h->td_a = 1.5;//0.155;
	par_h->td_b = 0.0;
	par_h->td_c = 0.0;
	//par_h->td_b = 4.95;
	//par_h->td_c = 2.01;
	par_h->boundCond = GlobalVariables::boundaryCondition;
	par_h->cb = 4182;
	par_h->k = 2;

	par_h->Ta = 25;
	par_h->defaultT = GlobalVariables::baseLineT;
	par_h->dt = 2.0;

	par_h->dx_2[0] = 0.0025;// 4.0 * 1e-6;
	par_h->dx_2[1] = 0.0025; //4.0 * 1e-6;
	par_h->dx_2[2] = 0.0025; //4.0 * 1e-6;

	for (int i = 0; i < BUFFER_HEATSOURCES; i++)
	{
		par_h->T_heat[i] = (float)GlobalVariables::baseLineT;
		par_h->heat_x[i] = 0;
		par_h->heat_y[i] = 0;
		par_h->heat_z[i] = 0;
	}
	par_h->N_heat = 0;
#if GPU
	updateParameters();
#endif
}
void PennesHeatTransfer_Gpu::setPerfusion(Matrix3D<bool>& _map)
{
	std::cout << "SetPerfusion " <<n<< std::endl;
	double w_b = 800 / (1e6 * 60) * 997 / (0.005 * 0.005 * 1 * 3.1415);
	
	for (int i = 0; i < n; i++)
	{
		if (_map.getData()[i] == false)
		{
			//wb_h->setValue(i, 0);
			wb_h->getData()[i] = 0;
		}
		else
		{
			//wb_h->setValue(i, w_b);
			wb_h->getData()[i] = w_b;
			//std::cout << "WB" << std::endl;
		}
		//if(wb_h->getData()[i] != 0)
		//	std::cout << " " << wb_h->getData()[i];
	}
#if GPU
	std::cout << "ersterErlfog" << std::endl;
	std::chrono::steady_clock::time_point begin_memcpy_malloc = std::chrono::steady_clock::now();
	checkErrorsCuda(hipMemcpy(wb_d, wb_h->getData(), sizeof(float)* n, hipMemcpyHostToDevice));
	std::chrono::steady_clock::time_point end_memcpy_malloc = std::chrono::steady_clock::now();
	std::cout << "memcpy: " << std::chrono::duration_cast<std::chrono::microseconds>(end_memcpy_malloc - begin_memcpy_malloc).count() << std::endl;
	hipDeviceSynchronize();
	checkLastCudaError("Kernel launch failed.");
#endif
}
void PennesHeatTransfer_Gpu::finiteStep(int _NrOfTimeSteps)
{
	//printParameters();
#if GPU
	finiteStep_GPU(_NrOfTimeSteps);
#else 
	finiteStep_CPU(_NrOfTimeSteps);
#endif 
}
//
//
#if GPU
void  PennesHeatTransfer_Gpu::finiteStep_GPU(int _NrOfTimeSteps)
{
	//std::cout << "finiteStep " << _NrOfTimeSteps << std::endl;
	std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
	
	updateParameters();
	//checkErrorsCuda(hipMemcpy(par_d, (void*)&par_h, sizeof(PennesEquationParameter), hipMemcpyHostToDevice));


	const int NUM_THREADS_PER_BLOCK = 32;//optimerung damit chache voll ausgef�llt (ist aber glaube ich lagnsamer)
	dim3 num_blocks, num_threads_per_block, number_of_threads;

	//xSweep
	number_of_threads.x = par_h->dim.y * par_h->dim.z;
	num_threads_per_block.x = NUM_THREADS_PER_BLOCK;
	num_blocks.x = number_of_threads.x / num_threads_per_block.x;
	if (0 != number_of_threads.x % num_threads_per_block.x) {
		num_blocks.x++;
	}
	/*std::cout << "X: num_blocks = " << num_blocks.x << " :: "
		<< "num_threads_per_block = " << num_threads_per_block.x << " :: num_of_threads: " << number_of_threads.x<< std::endl;*/

		//ySweep
	number_of_threads.y = par_h->dim.x * par_h->dim.z;
	num_threads_per_block.y = NUM_THREADS_PER_BLOCK;
	num_blocks.y = number_of_threads.y / num_threads_per_block.y;
	if (0 != number_of_threads.y % num_threads_per_block.y) {
		num_blocks.y++;
	}
	/*std::cout << "Y: num_blocks = " << num_blocks.y << " :: "
		<< "num_threads_per_block = " << num_threads_per_block.y << " :: num_of_threads: " << number_of_threads.y << std::endl;*/
		//zSweep
	number_of_threads.z = par_h->dim.x * par_h->dim.y;
	num_threads_per_block.z = NUM_THREADS_PER_BLOCK;
	num_blocks.z = number_of_threads.z / num_threads_per_block.z;
	if (0 != number_of_threads.z % num_threads_per_block.z) {
		num_blocks.z++;
	}
	/*std::cout << "Z: num_blocks = " << num_blocks.z << " :: "
		<< "num_threads_per_block = " << num_threads_per_block.z << " :: num_of_threads: " << number_of_threads.z << std::endl;*/

	for (int i = 0; i < _NrOfTimeSteps; i++)
	{


		//std::cout << "setHeatSources" << std::endl;
		setHeatSources << <1, par_h->N_heat >> > (T_d, par_d);
		hipDeviceSynchronize();
		checkLastCudaError("Kernel launch failed.");
		// MBTotal shared memory per block: 49152

		//Execute XSweep
		//std::cout << "Execute XSweep" << std::endl;
		//std::chrono::steady_clock::time_point begin_xSweep = std::chrono::steady_clock::now();

		int reservedSharedMemoryX = (num_threads_per_block.x * NUMBER_OF_ARRAYS * par_h->dim.x * sizeof(float));
		//std::cout << "sharedMemoryX: " << reservedSharedMemoryX << std::endl;;
		xSweep << <num_blocks.x, num_threads_per_block.x, reservedSharedMemoryX >> > (T_d, Ts_d, par_d);
		hipDeviceSynchronize();
		checkLastCudaError("Kernel launch failed.");
		/*std::chrono::steady_clock::time_point end_xSweep = std::chrono::steady_clock::now();
		std::cout << "xSweep_duration: " << std::chrono::duration_cast<std::chrono::microseconds>(end_xSweep - begin_xSweep).count() << std::endl;*/

		//Execute YSweep
		//std::cout << "Execute YSweep" << std::endl;
		//std::chrono::steady_clock::time_point begin_ySweep = std::chrono::steady_clock::now();
		int reservedSharedMemoryY = (num_threads_per_block.y * NUMBER_OF_ARRAYS * par_h->dim.y * sizeof(float));
		//std::cout << "sharedMemoryY: " << reservedSharedMemoryY << std::endl;;
		ySweep << <num_blocks.y, num_threads_per_block.y, reservedSharedMemoryY >> > (T_d, Ts_d, Tss_d, par_d);
		hipDeviceSynchronize();
		checkLastCudaError("Kernel launch failed.");
		/*std::chrono::steady_clock::time_point end_ySweep = std::chrono::steady_clock::now();
		std::cout << "ySweep_duration: " << std::chrono::duration_cast<std::chrono::microseconds>(end_ySweep - begin_ySweep).count() << std::endl;*/

		////Execute ZSweep
		//std::cout << "Execute ZSweep" << std::endl;
		//std::chrono::steady_clock::time_point begin_zSweep = std::chrono::steady_clock::now();
		int reservedSharedMemoryZ = (num_threads_per_block.z * NUMBER_OF_ARRAYS * par_h->dim.z * sizeof(float));
		//std::cout << "sharedMemoryZ: " << reservedSharedMemoryZ<< std::endl;;
		zSweep << <num_blocks.z, num_threads_per_block.z, reservedSharedMemoryZ >> > (T_d, Ts_d, Tss_d, Tnew_d, wb_d, par_d);
		hipDeviceSynchronize();
		checkLastCudaError("Kernel launch failed.");
		/*std::chrono::steady_clock::time_point end_zSweep = std::chrono::steady_clock::now();
		std::cout << "zSweep_duration: " << std::chrono::duration_cast<std::chrono::microseconds>(end_zSweep - begin_zSweep).count() << std::endl;*/

		setHeatSources << <1, par_h->N_heat >> > (T_d, par_d);
		hipDeviceSynchronize();
		checkLastCudaError("Kernel launch failed.");

		if (_NrOfTimeSteps > 1)
		{
			const int NUM_THREADS_PER_BLOCK = 1024;
			int num_of_threads = n;
			int num_threads_per_block = NUM_THREADS_PER_BLOCK;
			int num_blocks = num_of_threads / num_threads_per_block;
			if (n % num_threads_per_block != 0)
				num_blocks++;

			/*std::cout << "coypTnew2T: num_blocks = " << num_blocks << " :: "
				<< "num_threads_per_block = " << num_threads_per_block << " :: num_of_threads: " << num_of_threads << std::endl;*/

			copyT << < num_blocks, num_threads_per_block >> > (Tnew_d, T_d, n);
			hipDeviceSynchronize();
			checkLastCudaError("Kernel launch failed.");

		}



	}

	//DEBUG



	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
	//	std::cout << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << std::endl;// "finiteStep_duration: " <<
}
#endif
void  PennesHeatTransfer_Gpu::finiteStep_CPU(int _NrOfTimeSteps)
{
	//std::cout << "StepImlicitADI-Start" << std::endl;

	//crank-Nicolson - von nuemann adiabatisch randbedingung

	dim3 dim = { par_h->dim.x, par_h->dim.y, par_h->dim.z };
	float defaultValue = par_h->defaultT;
	float dt = par_h->dt;
	float dx_2[3] = { par_h->dx_2[0], par_h->dx_2[1], par_h->dx_2[2] };

	//std::cout << "T:mitte: " << (*P_n)[{dim.x / 2, dim[1] / 2, dim.z / 2}].T << std::endl;

	//Matrix3D<double>* T = new Matrix3D<double>(dim.x, dim[1], dim.z);

	
	Matrix3D<float> Ts_m = Matrix3D<float>(par_h->dim.x, par_h->dim.y, par_h->dim.z);
	Matrix3D<float> Tss_m = Matrix3D<float>(par_h->dim.x, par_h->dim.y, par_h->dim.z);

	
	if (GlobalVariables::boundaryCondition == 1)
	{
		T_h->setAmbientValue(defaultValue);
		Ts_m.setAmbientValue(defaultValue);
		Tss_m.setAmbientValue(defaultValue);
		
	}
	for (int t = 0; t < _NrOfTimeSteps; t++)
	{
		//Heating
	//std::cout << "Heating Temperatur: " << std::endl;
#pragma omp parallel for 
		for (int i = 0; i < par_h->N_heat; i++)
		{

			T_h->setValue({ par_h->heat_x[i],par_h->heat_y[i] ,par_h->heat_z[i] }, par_h->T_heat[i]);
		}
		//x-direction	

//#pragma omp parallel for collapse(2)
		for (int y = 0; y < dim.y; y++)
		{
			for (int z = 0; z < dim.z; z++)
			{


				float* a = new float[dim.x]();
				float* b = new float[dim.x]();
				float* c = new float[dim.x]();
				float* d = new float[dim.x]();

				float r1, r2, r3;
				float td; //thermal diffusivity (m^2/s)

				for (int x = 0; x < dim.x; x++)
				{



					//TODO k flux conservative 
						//TODO boundary conditition
					//td = (*P_n)[{i, j, k}].getK() / ((*P_n)[{i, j, k}].c* (*P_n)[{i, j, k}].rho);
					//td = (*P_n)[{i, j, k}].k / (*P_n)[{i, j, k}].M;
					td = getTd(par_h, (*T_h)[{x, y, z}]);
					r1 = r2 = r3 = td * dt / dx_2[0];

					a[x] = -r1 / 2.0;
					b[x] = 1.0 + r1;
					c[x] = -r1 / 2.0;


					d[x] = r1 / 2.0 * (*T_h)[{x - 1, y, z}] + r1 / 2.0 * (*T_h)[{x, y, z}]
						+ r2 * (*T_h)[{x, y - 1, z}] + r2 * (*T_h)[{x, y + 1, z}]
						+ r3 * (*T_h)[{x, y, z - 1}] + r3 * (*T_h)[{x, y, z + 1}]
						+ (1.0 - r1 - 2.0 * r2 - 2.0 * r3) * (*T_h)[{x, y, z}];

				}
				a[0] = 0;
				c[dim.x - 1] = 0;
				if (GlobalVariables::boundaryCondition == 0)//neumann -gleiche t_-1 = t_0
				{
					b[0] -= r1 / 2.0;
					b[dim.x - 1] -= r1 / 2.0;
				}
				else
				{

					d[0] += r1 / 2.0 * defaultValue;

					d[dim.x - 1] += r1 / 2.0 * defaultValue;
					/*d[0] += r3 / 2 * defaultValue + r2 / 2 * defaultValue +  r1 / 2 * defaultValue;
					d[par_h->dim.x - 1] += r3 / 2 * defaultValue + r2 / 2 * defaultValue +  r1 / 2 * defaultValue;*/
				}
				//d[0] = d[0] + a[0] * (*P_n)[{0, j, k}].T;
				//d[par_h->dim.x - 1] = d[par_h->dim.x - 1] + c[0] * (*P_n)[{par_h->dim.x - 1, j, k}].T;
				thomas(a, b, c, d, par_h->dim.x);

				for (int x = 0; x < par_h->dim.x; x++)
				{
				/*	if (d[x] < 20)
						std::cout << "dx :" <<x << " " << y << " " << z << " : " << d[x] << std::endl;*/
					(Ts_m)[{x, y, z}] = d[x];
				}
				delete[] a;
				delete[] b;
				delete[] c;
				delete[] d;
			}
		}





		//y-Direction

//#pragma omp parallel for collapse(2)
		for (int x = 0; x < par_h->dim.x; x++)
		{
			for (int z = 0; z < par_h->dim.z; z++)
			{
				float* a = new float[par_h->dim.y]();
				float* b = new float[par_h->dim.y]();
				float* c = new float[par_h->dim.y]();
				float* d = new float[par_h->dim.y]();
				float r1, r2, r3;
				float td; //thermal diffusivity (m^2/s)

				for (int y = 0; y < par_h->dim.y; y++)
				{

					//td = (*P_n)[{i, j, k}].getK() / ((*P_n)[{i, j, k}].c* (*P_n)[{i, j, k}].rho);
					//td = (*P_n)[{i, j, k}].k / (*P_n)[{i, j, k}].M;
					td = getTd(par_h, (*T_h)[{x, y, z}]);
					r1 = r2 = r3 = td * dt / dx_2[1];

					a[y] = -r2 / 2.0;
					b[y] = 1.0 + r2;
					c[y] = -r2 / 2.0;

					d[y] = r1 / 2.0 * (*T_h)[{x - 1, y, z}] + r1 / 2.0 * (*T_h)[{x + 1, y, z}]
						+ r1 / 2.0 * (Ts_m)[{x - 1, y, z}] + r1 / 2.0 * (Ts_m)[{x + 1, y, z}]
						+ r2 / 2.0 * (*T_h)[{x, y - 1, z}] + r2 / 2.0 * (*T_h)[{x, y + 1, z}]
						+ r3 * (*T_h)[{x, y, z - 1}] + r3 * (*T_h)[{x, y, z + 1}]
						- r1 * (Ts_m)[{x, y, z}]
						+ (1.0 - r1 - r2 - 2.0 * r3) * (*T_h)[{x, y, z}];
				}
				a[0] = 0;
				c[par_h->dim.y - 1] = 0;
				//cout << "d[0]_y " << d[0] << std::endl << std::endl;
				if (GlobalVariables::boundaryCondition == 0)//neumann -gleiche t_-1 = t_0
				{
					b[0] -= r2 / 2.0;
					b[par_h->dim.y - 1] -= r2 / 2.0;
				}
				else
				{
					d[0] += r2 / 2.0 * defaultValue;
					d[par_h->dim.y - 1] += r2 / 2.0 * defaultValue;
					/*d[0] += r3 / 2 * defaultValue + r2 / 2 * defaultValue + 2 * r1 / 2 * defaultValue;
					d[par_h->dim.y - 1] += r3 / 2 * defaultValue + r2 / 2 * defaultValue + 2 * r1 / 2 * defaultValue;*/
				}


				thomas(a, b, c, d, par_h->dim.y);
				//cout << "d[0]_y " << d[0] << std::endl << std::endl;

				for (int y = 0; y < par_h->dim.y; y++)
				{
					/*if (d[l] < 20)
						std::cout << "dy :" << i << " " << l << " " << k << " : " << d[l] << std::endl;*/
						/*if ((*P_n)[{i, l, k}].keepTconstant)
							(*T_ss)[{i, l, k}] = (*T_s)[{i, l, k}];
						else*/
					(Tss_m)[{x, y, z}] = d[y];
				}
				delete[] a;
				delete[] b;
				delete[] c;
				delete[] d;
			}
		}



		//z.direction
//#pragma omp parallel for collapse(2)
		for (int x = 0; x < par_h->dim.x; x++)
		{
			for (int y = 0; y < par_h->dim.y; y++)
			{
				float* a = new float[par_h->dim.z]();
				float* b = new float[par_h->dim.z]();
				float* c = new float[par_h->dim.z]();
				float* d = new float[par_h->dim.z]();
				float r1, r2, r3;
				float td; //thermal diffusivity (m^2/s)
				float res;

				for (int z = 0; z < par_h->dim.z; z++)
				{

					//td = (*P_n)[{i, j, k}].getK() / ((*P_n)[{i, j, k}].c* (*P_n)[{i, j, k}].rho);
					//td = (*P_n)[{i, j, k}].k / (*P_n)[{i, j, k}].M;
					td = getTd(par_h, (*T_h)[{x, y, z}]);
					r1 = r2 = r3 = td * dt / dx_2[2];

					a[z] = -r3 / 2.0;
					b[z] = 1.0 + r3;
					c[z] = -r3 / 2.0;

					d[z] = r1 / 2.0 * (*T_h)[{x - 1, y, z}] + r1 / 2.0 * (*T_h)[{x + 1, y, z}]
						+ r1 / 2.0 * (Ts_m)[{x - 1, y, z}] + r1 / 2.0 * (Ts_m)[{x + 1, y, z}]
						+ r2 / 2.0 * (*T_h)[{x, y - 1, z}] + r2 / 2.0 * (*T_h)[{x, y + 1, z}]
						+ r2 / 2.0 * (Tss_m)[{x, y - 1, z}] + r2 / 2.0 * (Tss_m)[{x, y + 1, z}]
						+ r3 / 2.0 * (*T_h)[{x, y, z - 1}] + r3 / 2.0 * (*T_h)[{x, y, z + 1}]
						- r1 * (Ts_m)[{x, y, z}] - r2 * (Tss_m)[{x, y, z}]
						+ (1.0 - r1 - r2 - r3) * (*T_h)[{x, y, z}];
					/*		d[k] = r1 / 2 * (*T)[{i - 1, j, k}] + r1 / 2 * (*T)[{i + 1, j, k}]
								+ r1 / 2 * (*T_s)[{i - 1, j, k}] + r1 / 2 * (*T_s)[{i + 1, j, k}]
								+ r2 / 2 * (*T)[{i, j - 1, k}] + r2 / 2 * (*T)[{i, j + 1, k}]
								+ r2 / 2 * (*T_ss)[{i, j - 1, k}] + r2 / 2 * (*T_ss)[{i, j + 1, k}]
								+ r3 / 2 * (*T)[{i, j, k - 1}] + r3 / 2 * (*T)[{i, j, k + 1}]
								- r1 * (*T_s)[{i, j, k}] - r2 * (*T_ss)[{i, j, k}]
								+ (1 - r1 - r2 - r3) * (*T)[{i, j, k}];*/

				}
				//cout <<"d[0]_z "<< d[0]<<std::endl;
				a[0] = 0;
				c[par_h->dim.z - 1] = 0;

				if (GlobalVariables::boundaryCondition == 0)//neumann -gleiche t_-1 = t_0
				{

					b[0] -= r3 / 2.0;
					b[par_h->dim.z - 1] -= r3 / 2.0;
				}
				else
				{
					d[0] += r3 / 2.0 * defaultValue;
					//cout << "d[0] " << d[0] << std::endl;
					d[par_h->dim.z - 1] += r3 / 2.0 * defaultValue;
					/*d[0] += r3 / 2 * defaultValue + 2 * r2 / 2 * defaultValue + 2 * r1 / 2 * defaultValue;
					d[par_h->dim.z-1] += r3 / 2 * defaultValue + 2 * r2 / 2 * defaultValue + 2 * r1 / 2 * defaultValue;*/
				}


				//cout << "d[0] " << d[0] << std::endl;
				thomas(a, b, c, d, par_h->dim.z);
				//cout << "d[0]_z " << d[0] << std::endl << std::endl;
				float P_t;
				for (int z = 0; z < par_h->dim.z; z++)
				{

					//if (d[z] > 100 || d[z] < 10)
					//{
					//	std::cout << "Falsch d pen " <<x<<" "<<y<<" "<<z<<" " << d[z]<<std::endl;
					//}
					/*if(hf_t > 0.1|| hf_t < -0.1)
						std::cout  <<"hft_t: " <<hf_t << std::endl;*/
						/*if (d[l] < 20)
							std::cout<<"d :" << i << " " << j << " " << l <<" : "<<d[l]<<  std::endl;*/
							/*if (hf_t> 0)
								std::cout <<"hft: "<<hf_t<<" " << i << " " << j << " " << l << std::endl;*/
								/*if (d[l] > 100)
									std::cout << i<< " " << j << " " <<l << std::endl;*/
									//int v;
									//if (v_n.isVessel)
									//{
									//	v = 1;
									//	//std::cout << "isVessel" << std::endl;
									//}
									//	
									//else
									//	v = 0;
									//Pt = rho_b * w * Cb / (rho * Ct) * (T_amb - T_new)
					//float P_t = v_n.w_b * v_n.c_b * (v_n.T_a - d[l]) * (v_n.getTd() / v_n.k);
					/*if (P_t > 0)
						std::cout <<"P_t" << P_t << " Diff "<< (v_n.T_a - d[l])<<std::endl;*/
						//res = d[l] + dt * P_t;//+ hf_t   /((v_n.getTd()/v_n.k))
						//+dt * (v_n.w_b * v_n.c_b * (v_n.T_a - v_n.T))
						/*if (v_n.keepTconstant == true)
						{
							(*P_new)[{i, j, l}].T = v_n.T;
							(*T_new)[{i, j, l}] = v_n.T;

						}
						else*/
					
					P_t = (*wb_h)[{x, y, z}] * par_h->cb * (par_h->Ta - d[z]) *(getTd(par_h, (*T_h)[{x, y, z}]) / par_h->k);
						(*Tnew_h)[{x, y, z}] = d[z] + dt * P_t;
					

				}
				delete[] a;
				delete[] b;
				delete[] c;
				delete[] d;
			}
		}


		long float Heatsum = 0;

		if (_NrOfTimeSteps > 1)
		{
		//	std::cout << "multi" << std::endl;

			for (int i = 0; i < par_h->dim.x; i++)
			{
				for (int j = 0; j < par_h->dim.y; j++)
				{
					for (int k = 0; k < par_h->dim.z; k++)
					{

						(*T_h)[{i, j, k}] = (*Tnew_h)[{i, j, k}];

						Heatsum += (*T_h)[{i, j, k}];
						//if ((*P_new)[{i, j, k}].T <= 20 || (*P_new)[{i, j, k}].T >= 22)
						//{
						//	std::cout <<"null: " << i << " " << " " << j << " " << k <<" "<< (*P_new)[{i, j, k}].T << std::endl;
						//}

					}
				}
			}
		}

		//debugViewer(T_s);
		//ParameterOptimization::debugViewer(T_ss);
		//ParameterOptimization::debugViewer(P_new);


		//delete T;
		/*delete T_s;
		delete T_ss;
		delete T_new;*/
		//std::cout << "stepImplicitADI - Heatsum: "<<Heatsum<<" " << (double)(Heatsum / (P_n->getpar_h->dimensions()[0] * P_n->getpar_h->dimensions()[1] * P_n->getDimensions()[2])) << std::endl;
		//debugViewer(P_new);
		//std::cout << "stepImplicitADI-End" << std::endl;
	}
	



}

void PennesHeatTransfer_Gpu::printParameters()
{
	std::cout<< "dx_2 " << par_h->dx_2[0] << " " << par_h->dx_2[1] << " " << par_h->dx_2[2] << std::endl;
	std::cout << "dt " << par_h->dt << std::endl;
	std::cout << "td_a " << par_h->td_a << std::endl;
	std::cout << "defT " << par_h->defaultT << std::endl;
	std::cout << "bound " << par_h->boundCond << std::endl;
	std::cout << "dim " << par_h->dim.x<<" "<< par_h->dim.y<<" "<<par_h->dim.z << std::endl;
	std::cout << "N_heat " << par_h->N_heat << std::endl;
	for (int i = 0; i < par_h->N_heat; i++)
	{
		std::cout << par_h->T_heat[i] << " "<< par_h->heat_x[i]<<" "<< par_h->heat_y[i] << " " << par_h->heat_z[i] << std::endl;
	}
	
}



//wenn man mehere zeitschrritte will
//fkt(--int tsteps)
//atomic int currentstep
// while(current step < tsptes)
// {
// syncthreads
// currentstep++
// }
//