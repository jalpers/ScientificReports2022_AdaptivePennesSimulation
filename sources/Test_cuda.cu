#include "hip/hip_runtime.h"
//cudaWorker.cu




#include "Test_cuda.h"
#include<cuda_util.h>
#include<TestClass.h>
//__global__ void kernel1(float deviceData[])
//
//{
//
//}
__global__ void cuda_hello(A a) {//float* _T_d, float* _Ts_d
	printf("Hello World from GPU!\n");
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	printf("tid %d: \n", tid);
	a.increment();
	a.print_data();
}
void Test_Cuda::doCuda()

{
	std::cout << "testCuda" << std::endl;
	int handle = 0;
	bool ft = InitDevice(handle);
	std::cout << (ft ? "True" : "False") << std::endl;
	const int n = 1000;
	float a[n];
	float b[n];
	float* a_d, * b_d;
	for (int i = 0; i < n; i++)
	{
		a[i] = 10;
		b[i] = 1;
	}
	//checkErrorsCuda(hipMalloc(((void**)&a_d, n * sizeof(float)));

	/*checkErrorsCuda(hipMalloc(((void**)&b_d, n * sizeof(float)));

	checkErrorsCuda(hipMemcpy(a_d, a, hipMemcpyHostToDevice));*/

	A h_a;
	h_a.increment();
	h_a.print_data();
	cuda_hello << <1, 1 >> > (h_a);
	hipDeviceSynchronize();
	// 
	//unpack the vector into an array

	//use CUDA functions to to memcpy data and to launch a kernel

	//kernel1 << <64, 64 >> > (deviceData); //launching kernel legal, it's a .cu file

}
Test_Cuda::Test_Cuda()
{

}
Test_Cuda::~Test_Cuda()
{

}
